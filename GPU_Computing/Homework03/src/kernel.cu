#include "hip/hip_runtime.h"
/*************************************************************************************************
 *
 *        Computer Engineering Group, Heidelberg University - GPU Computing Exercise 03
 *
 *                           Group : TBD
 *
 *                            File : main.cu
 *
 *                         Purpose : Memory Operations Benchmark
 *
 *************************************************************************************************/

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chCommandLine.h>
#include <chTimer.hpp>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

//
// Kernels
//

__global__ void 
globalMemCoalescedKernel(int* out, const int* in, int size_in_bytes)
{
    int num_kernels = blockDim.x * gridDim.x;

    int size = size_in_bytes / sizeof(int);

    int copies_per_kernel = size + num_kernels - 1 / num_kernels;

    for (int i = 0; i < copies_per_kernel; i++){
        int index =  blockIdx.x * blockDim.x + threadIdx.x + i * num_kernels;
        if (index < size){
            //out[index] = 1;
        }
    }
}

void 
globalMemCoalescedKernel_Wrapper(dim3 gridDim, dim3 blockDim, int* out, const int* in, int size_in_bytes) {
	globalMemCoalescedKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( out, in, size_in_bytes );
}

__global__ void 
globalMemStrideKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemStrideKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemStrideKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

__global__ void 
globalMemOffsetKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemOffsetKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemOffsetKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

