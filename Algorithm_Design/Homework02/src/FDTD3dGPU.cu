#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "FDTD3dGPU.h"

#include <iostream>
#include <algorithm>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "FDTD3dGPUKernel.cuh"

bool getTargetDeviceGlobalMemSize(memsize_t *result, const int argc,
                                  const char **argv) {
  int deviceCount = 0;
  int targetDevice = 0;
  size_t memsize = 0;

  // Get the number of CUDA enabled GPU devices
  printf(" hipGetDeviceCount\n");
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, (const char **)argv);

  // Query target device for maximum memory allocation
  printf(" hipGetDeviceProperties\n");
  struct hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

  memsize = deviceProp.totalGlobalMem;

  // Save the result
  *result = (memsize_t)memsize;
  return true;
}

bool fdtdGPU(float *output, const float *input, const float *coeff,
             const int dimx, const int dimy, const int dimz, const int radius,
             const int timesteps, const int argc, const char **argv) {
  const int outerDimx = dimx + 2 * radius;
  const int outerDimy = dimy + 2 * radius;
  const int outerDimz = dimz + 2 * radius;
  const size_t volumeSize = outerDimx * outerDimy * outerDimz;
  int deviceCount = 0;
  int targetDevice = 0;
  float *bufferOut = 0;
  float *bufferIn = 0;
  dim3 dimBlock;
  dim3 dimGrid;

  int boundary = 128;
  if (checkCmdLineFlag(argc, argv, "boundary")) {
    boundary = getCmdLineArgumentInt(argc, argv, "boundary");
  }
  printf("Boundary: %i\n", boundary);

  // Ensure that the inner data starts on a 128B boundary
  const int padding = (boundary / sizeof(float)) - radius;
  const size_t paddedVolumeSize = volumeSize + padding;

#ifdef GPU_PROFILING
  hipEvent_t profileStart = 0;
  hipEvent_t profileEnd = 0;
  const int profileTimesteps = timesteps - 1;

  if (profileTimesteps < 1) {
    printf(
        " cannot profile with fewer than two timesteps (timesteps=%d), "
        "profiling is disabled.\n",
        timesteps);
  }

#endif

  // Check the radius is valid
  if (radius > 10 || radius < 1) {
    printf("radius is invalid, must be between 1 and 10.\n");
    exit(EXIT_FAILURE);
  }
  else{
    printf("Radius: %i\n", radius);
  }

  // Get the number of CUDA enabled GPU devices
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipSetDevice(targetDevice));

  // Allocate memory buffers
  checkCudaErrors(
      hipMalloc((void **)&bufferOut, paddedVolumeSize * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&bufferIn, paddedVolumeSize * sizeof(float)));

  // Check for a command-line specified block size
  int userBlockSize;

  if (checkCmdLineFlag(argc, (const char **)argv, "block-size")) {
    userBlockSize = getCmdLineArgumentInt(argc, argv, "block-size");
    // Constrain to a multiple of k_blockDimX
    userBlockSize = (userBlockSize / k_blockDimX * k_blockDimX);

    // Constrain within allowed bounds
    userBlockSize = MIN(MAX(userBlockSize, k_blockSizeMin), k_blockSizeMax);
  } else {
    userBlockSize = k_blockSizeMax;
  }

  // Check the device limit on the number of threads
  struct hipFuncAttributes funcAttrib;
  checkCudaErrors(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(FiniteDifferencesKernel<10>)));

  userBlockSize = MIN(userBlockSize, funcAttrib.maxThreadsPerBlock);

  // Set the block size
  dimBlock.x = k_blockDimX;
  // Visual Studio 2005 does not like std::min
  //    dimBlock.y = std::min<size_t>(userBlockSize / k_blockDimX,
  //    (size_t)k_blockDimMaxY);
  dimBlock.y = ((userBlockSize / k_blockDimX) < (size_t)k_blockDimMaxY)
                   ? (userBlockSize / k_blockDimX)
                   : (size_t)k_blockDimMaxY;
  dimGrid.x = (unsigned int)ceil((float)dimx / dimBlock.x);
  dimGrid.y = (unsigned int)ceil((float)dimy / dimBlock.y);
  printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
  printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);

  // Check the block size is valid
  if (dimBlock.x < radius || dimBlock.y < radius) {
    printf("invalid block size, x (%d) and y (%d) must be >= radius (%d).\n",
           dimBlock.x, dimBlock.y, radius);
    exit(EXIT_FAILURE);
  }

  // Copy the input to the device input buffer
  checkCudaErrors(hipMemcpy(bufferIn + padding, input,
                             volumeSize * sizeof(float),
                             hipMemcpyHostToDevice));

  // Copy the input to the device output buffer (actually only need the halo)
  checkCudaErrors(hipMemcpy(bufferOut + padding, input,
                             volumeSize * sizeof(float),
                             hipMemcpyHostToDevice));

  // Copy the coefficients to the device coefficient buffer
  checkCudaErrors(
      hipMemcpyToSymbol(HIP_SYMBOL(stencil), (void *)coeff, (radius + 1) * sizeof(float)));

#ifdef GPU_PROFILING

  // Create the events
  checkCudaErrors(hipEventCreate(&profileStart));
  checkCudaErrors(hipEventCreate(&profileEnd));

#endif

  // Execute the FDTD
  float *bufferSrc = bufferIn + padding;
  float *bufferDst = bufferOut + padding;
  printf(" GPU FDTD loop\n");

#ifdef GPU_PROFILING
  // Enqueue start event
  checkCudaErrors(hipEventRecord(profileStart, 0));
#endif

  for (int it = 0; it < timesteps; it++) {
    printf("\tt = %d ", it);

    // Launch the kernel
    printf("launch kernel\n");

    switch (radius){
      case 1:
        FiniteDifferencesKernel<1><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 2:
        FiniteDifferencesKernel<2><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 3:
        FiniteDifferencesKernel<3><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 4:
        FiniteDifferencesKernel<4><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 5:
        FiniteDifferencesKernel<5><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 6:
        FiniteDifferencesKernel<6><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 7:
        FiniteDifferencesKernel<7><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 8:
        FiniteDifferencesKernel<8><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 9:
        FiniteDifferencesKernel<9><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
      case 10:
        FiniteDifferencesKernel<10><<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx,
                                                   dimy, dimz);
        break;
    }

    // Toggle the buffers
    // Visual Studio 2005 does not like std::swap
    //    std::swap<float *>(bufferSrc, bufferDst);
    float *tmp = bufferDst;
    bufferDst = bufferSrc;
    bufferSrc = tmp;
  }

  printf("\n");

#ifdef GPU_PROFILING
  // Enqueue end event
  checkCudaErrors(hipEventRecord(profileEnd, 0));
#endif

  // Wait for the kernel to complete
  checkCudaErrors(hipDeviceSynchronize());

  // Read the result back, result is in bufferSrc (after final toggle)
  checkCudaErrors(hipMemcpy(output, bufferSrc, volumeSize * sizeof(float),
                             hipMemcpyDeviceToHost));

// Report time
#ifdef GPU_PROFILING
  float elapsedTimeMS = 0;

  if (profileTimesteps > 0) {
    checkCudaErrors(
        hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd));
  }

  if (profileTimesteps > 0) {
    // Convert milliseconds to seconds
    double elapsedTime = elapsedTimeMS * 1.0e-3;
    double avgElapsedTime = elapsedTime / (double)profileTimesteps;
    // Determine number of computations per timestep
    size_t pointsComputed = dimx * dimy * dimz;
    // Determine throughput
    double throughputM = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
    printf(
        "FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %u Points, "
        "NumDevsUsed = %u, Blocksize = %u\n",
        throughputM, avgElapsedTime, pointsComputed, 1,
        dimBlock.x * dimBlock.y);
  }

#endif

  // Cleanup
  if (bufferIn) {
    checkCudaErrors(hipFree(bufferIn));
  }

  if (bufferOut) {
    checkCudaErrors(hipFree(bufferOut));
  }

#ifdef GPU_PROFILING

  if (profileStart) {
    checkCudaErrors(hipEventDestroy(profileStart));
  }

  if (profileEnd) {
    checkCudaErrors(hipEventDestroy(profileEnd));
  }

#endif
  return true;
}
